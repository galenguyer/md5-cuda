#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>
#include "md5.cu"

int main() {
    char* h_str = "a";
    char* d_str;
    unsigned char* h_res = (unsigned char*)malloc(sizeof(unsigned char)*(32 + 1));
    unsigned char* d_res;
    hipMalloc((void**)&d_str, sizeof(char));
    hipMalloc((void**)&d_res, sizeof(char)* 32);
    hipMemcpy(d_str, h_str, sizeof(char), hipMemcpyHostToDevice);

    md5<<<1, 1>>>(d_str, (uint32_t)strlen(h_str), d_res);

    hipMemcpy(h_res, d_res, sizeof(unsigned char)*(32), hipMemcpyDeviceToHost);

    hipFree(d_str);
    hipFree(d_res);

    char* res = (char*)malloc(sizeof(char)*32);
    for (int i = 0; i < 16; i++) {
        sprintf(&res[i*2], "%2.2x", h_res[i]);
    }
    
    puts(res);
}