#include <hip/hip_runtime.h>
#include <stdint.h>
#include "md5.cu"

int main() {
    char* h_str = "a";
    char* d_str;
    char* h_res = (char*)malloc(sizeof(char)*(32 + 1));
    char* d_res;
    hipMalloc((void**)&d_str, sizeof(char));
    hipMalloc((void**)&d_res, sizeof(char)* 32);
    hipMemcpy(d_str, h_str, sizeof(char), hipMemcpyHostToDevice);

    md5<<<1, 1>>>(d_str, (uint32_t)sizeof(h_str), d_res);

    hipMemcpy(h_res, d_res, sizeof(char)*(32), hipMemcpyDeviceToHost);

    hipFree(d_str);
    hipFree(d_res);

    for (int i = 0; i < 32; i++) {
        printf("%2.2x", h_res[i]);
    }
    puts("");
};